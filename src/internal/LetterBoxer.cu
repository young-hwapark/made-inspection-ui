#include "hip/hip_runtime.h"
#include "LetterBoxer.h"
#include <hip/hip_runtime.h>
#include <iostream>

LetterBoxer::LetterBoxer(int bufferWidth, int bufferHeight, int embedWidth, int embedHeight, int channels):
    pylonWidth(bufferWidth),
    pylonHeight(bufferHeight),
    bufferSize(bufferWidth * bufferHeight * channels),
    bufferChannels(channels),
    darknetWidth(embedWidth),
    darknetHeight(embedHeight),
    darknetSize(darknetWidth * darknetHeight * channels),
    resizeFactor(double(pylonWidth) / double(darknetWidth)),
    embeddedImageSize(darknetWidth * darknetHeight),
    embeddedImageHeight(int(pylonHeight / resizeFactor)),
    blockSize(256),
    numBlocks((blockSize + bufferSize - 1) / blockSize),
    padding(((darknetHeight - embeddedImageHeight) / 2) * pylonWidth),
    smallpadding(((darknetHeight - embeddedImageHeight) / 2) * darknetWidth)
{
    hipMallocManaged(&charBuffer, bufferSize * sizeof(unsigned char));
    hipMallocManaged(&floatBuffer, bufferSize * sizeof(float));
    hipMallocManaged(&smallBuffer, darknetSize * sizeof(float));
}

LetterBoxer::~LetterBoxer()
{
    hipDeviceSynchronize();
    hipFree(charBuffer);
    hipFree(floatBuffer);
    hipFree(smallBuffer);
}

__global__ void initialFloat(float *buffer, int size)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < size; i += stride){
        buffer[i] = 0.5f;
    }
}

__global__ void embedLetterbox(float *bufferOut, unsigned char *bufferIn, int embeddedImageSize, int smallsize, int bufferSize, int channels, int embeddedImageWidth, int bufferWidth, double factor, int padding)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < embeddedImageSize * 3; i += stride){
        int j = i % embeddedImageSize;
        bufferOut[(i / embeddedImageSize) * smallsize + j + padding] =
                bufferIn[(i / embeddedImageSize) * (bufferSize / 3) + int(double(j / embeddedImageWidth) * factor) * bufferWidth + int(double(j % embeddedImageWidth) * factor)] / 255.0;
    }
}

void LetterBoxer::letterbox(unsigned char *pylonBuffer, float *darknetBuffer)
{
    hipMemcpy(charBuffer, pylonBuffer, bufferSize * sizeof(unsigned char), hipMemcpyHostToDevice);

    initialFloat<<<numBlocks, blockSize>>>(smallBuffer, darknetSize);
    embedLetterbox<<<numBlocks, blockSize>>>(smallBuffer, charBuffer, embeddedImageHeight * darknetWidth, embeddedImageSize, bufferSize, bufferChannels, darknetWidth, pylonWidth, resizeFactor, smallpadding);

    hipDeviceSynchronize();

    hipMemcpy(darknetBuffer, smallBuffer, darknetSize * sizeof(float), hipMemcpyDeviceToHost);
}
